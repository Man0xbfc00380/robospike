#include "hip/hip_runtime.h"
#include "exp/exp0_cuda.cuh"
#include "hip/hip_runtime.h"
#include ""

template<typename T>
__global__ void matAdd_cuda(T *a,T *b,T *sum)
{
	int i = blockIdx.x*blockDim.x+ threadIdx.x;
    sum[i] = a[i] + b[i];
}

float* matAdd(float *a,float *b,int length)
{
    int device = 5;
    hipSetDevice(device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    int threadMaxSize = devProp.maxThreadsPerBlock;
    int blockSize = (length+threadMaxSize-1)/threadMaxSize;
    dim3 thread(threadMaxSize);
    dim3 block(blockSize);
    int size = length * sizeof(float);
    float *sum =(float *)malloc(size);
    float *sumGPU,*aGPU,*bGPU;
    hipMalloc((void**)&sumGPU,size);
    hipMalloc((void**)&aGPU,size);
    hipMalloc((void**)&bGPU,size);
    hipMemcpy((void*)aGPU,(void*)a,size,hipMemcpyHostToDevice);
    hipMemcpy((void*)bGPU,(void*)b,size,hipMemcpyHostToDevice);
    matAdd_cuda<float><<<block,thread>>>(aGPU,bGPU,sumGPU);
    hipMemcpy(sum,sumGPU,size,hipMemcpyDeviceToHost);
    hipFree(sumGPU);
    hipFree(aGPU);
    hipFree(bGPU);
    return sum;
}

__global__ void kernel(int* a, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int item = a[idx];
        for (int i = 0; i < 100; i++) {
            a[idx] += item;
            
            // ^^^^^ Extra code to evaluate time
            if (idx > 0) {
                a[idx] += a[idx - 1];
            }

            // ^^^^^ Results
            // open: 
            // [INFO] [Regular_callback11]: [PID: 2176175] [kernelCpp] [s: 0] [us: 34]
            // [INFO] [Regular_callback11]: [PID: 2176175] [hipStreamSynchronize] [s: 0] [us: 18117]
            // close:
            // [INFO] [Regular_callback11]: [PID: 2178310] [kernelCpp] [s: 0] [us: 43]
            // [INFO] [Regular_callback11]: [PID: 2178310] [hipStreamSynchronize] [s: 0] [us: 1992]
        }
        a[idx] += 2;
    }
}

void kernelCpp(int* h_a, int N, int threadsPerBlock, int blocksPerGrid,
            int* d_a, hipStream_t stream1)
{
    kernel<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(d_a, N);
    kernel<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(d_a, N);
}

